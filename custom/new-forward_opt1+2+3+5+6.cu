#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"
#include "hip/hip_fp16.h"

#define T_WIDTH 32
#define BT_WIDTH 8

__constant__ __half k_half[16*4*7*7];
__constant__ float k[16*4*7*7];

__global__ void conv_forward_kernel(float * __restrict__ y, const float * __restrict__ x, const int B, const int M, const int C, const int H, const int W, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function parameter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int m = blockIdx.z;
    int b = blockIdx.y * blockDim.y + threadIdx.y;
    int coor = blockIdx.x * blockDim.x + threadIdx.x;
    int w = coor % W_out;
    int h = coor / W_out;
    float output = 0.0;

    if ((h < H_out) && (w < W_out) && b < B) {
        __half2 temp(0.0f, 0.0f);
        
        #pragma unroll(4)
        for (int c = 0; c < C; c++) {
            
            #pragma unroll(7)
            for (int p = 0; p < K; p++) {
                #pragma unroll(3)
                for (int q = 0; q < K/2; q++) {
                    __half x_half = __float2half( x4d(b, c, h+p,w+2*q) );
                    __half x_half_1 = __float2half( x4d(b, c, h+p,w+2*q+1) );
                    __half k_half = __float2half( k4d(m, c, p, 2*q) );
                    __half k_half_1 = __float2half( k4d(m, c, p, 2*q+1) );
                    temp = __hadd2(__hmul2(__halves2half2(x_half,x_half_1),__halves2half2(k_half,k_half_1)),temp);
                }
            
                output += x4d(b, c, h+p,w+K-1)* k4d(m, c, p, K-1);
            }
        }
	
	    output += __half2float(__hadd(__high2half(temp), __low2half(temp)));
	    
        y4d(b, m, h, w) = output;
    }

#undef y4d
#undef x4d
#undef k4d
}

__host__ void conv_forward_gpu(float *host_y, const float *host_x, const float *host_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Declare relevant device pointers
    float* device_y;
    float* device_x;
    //float* device_k;

    // Allocate memory and copy over the relevant data structures to the GPU
    hipMalloc((void**)(&device_y), B * M * (H-K+1)*(W-K+1)*sizeof(float));
    hipMalloc((void**)(&device_x), B * C * H * W * sizeof(float));
    //hipMalloc((void**)(&device_k), M * C * K * K *sizeof(float));

    hipMemcpy(device_x, host_x,  B * C * H * W * sizeof(float), hipMemcpyHostToDevice);
    //hipMemcpy(device_k, host_k, M * C * K * K *sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(k), host_k, M * C * K * K *sizeof(float));

    // Set the kernel dimensions and call the kernel
    dim3 dimGrid(ceil((1.0*(W-K+1)*(H-K+1))/T_WIDTH), ceil((1.0*B)/BT_WIDTH), M);
    dim3 dimBlock(T_WIDTH, BT_WIDTH, 1);

    conv_forward_kernel<<<dimGrid,dimBlock>>>(device_y, device_x, B, M, C, H, W, K);


    // Copy the output back to host
    hipMemcpy(host_y, device_y, B * M * (H-K+1)*(W-K+1)*sizeof(float), hipMemcpyDeviceToHost);
    


    // Free device memory
    hipFree(device_y);
    hipFree(device_x);
    //hipFree(device_k);

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
}

__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
